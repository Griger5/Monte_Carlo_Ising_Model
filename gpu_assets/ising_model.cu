
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

__device__ void switchSpin(int *grid, int *cols, int i, int j) {
    grid[i*(*cols) + j] = -1 * grid[i*(*cols) + j];
}

__device__ void calculateEnergy(int *grid, int *cols, int i, int j, int *energy) {
    *energy = -grid[i*(*cols) + j] * (grid[(i+1)*(*cols) + j] + grid[(i-1)*(*cols) + j] + grid[i*(*cols) + (j+1)] + grid[i*(*cols) + (j-1)]);
}

extern "C"
__global__ void runIsingModel(int *grid, int *rows, int *cols, double *temp, int *steps) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = gridDim.x * blockDim.x;

    hiprandState_t rng;
	hiprand_init(clock64(), index, 0, &rng);

    int i, j, energy;
    double r;
    
    for (int k = index; k < *steps; k+=stride) {
        i = hiprand_uniform_double(&rng) * (*rows-2) + 1;
        i = (int)(i+0.5);
        j = hiprand_uniform_double(&rng) * (*cols-2) + 1;
        j = (int)(j+0.5);

        calculateEnergy(grid, cols, i, j, &energy);

        if (energy > 0) {
            switchSpin(grid, cols, i, j);
        }
        else if (energy < 0) {
            r = hiprand_uniform_double(&rng);
            if (r < exp(2*energy/(*temp))) {
                switchSpin(grid, cols, i, j);
            }
        }
    }
}